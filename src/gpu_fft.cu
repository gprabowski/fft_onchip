#include "hip/hip_runtime.h"
#include <algorithm>
#include <random>
#include <vector>

#include <config.hpp>
#include <perf_test.cuh>

#include <common.cuh>
#include <reference.cuh>
#include <tensor_fft_64.cuh>
#include <testing.cuh>

int main() {
  using config::CT;
  using config::N;

  std::random_device rd;
  std::uniform_real_distribution<float> dist(0.0, 1.0);

  std::vector<CT> data(N);

  // generate data
  std::transform(begin(data), end(data), begin(data), [&](auto) {
    return CT{dist(rd), dist(rd)};
  });

  // compare correctness
  std::vector<config::CT> out_algorithm, out_reference;

  using customExec = fft::tensor_fft_64<config::CT, config::N, 4, 2>;
  using refExec = fft::reference_fft<config::N>;

  auto alg_data = data;
  auto ref_data = data;

  const auto alg_run_transfers =
      testing::run_perf_and_corr_tests<config::CT, config::N, customExec, true>(
          alg_data, out_algorithm);

  const auto alg_run_no_transfers =
      testing::run_perf_and_corr_tests<config::CT, config::N, customExec,
                                       false>(alg_data, out_algorithm);

  const auto ref_run_transfers =
      testing::run_perf_and_corr_tests<refExec::VT, config::N, refExec, true>(
          ref_data, out_reference);

  const auto ref_run_no_transfers =
      testing::run_perf_and_corr_tests<refExec::VT, config::N, refExec, false>(
          ref_data, out_reference);

  double mse{0.0};

  for (int i = 0; i < out_reference.size(); ++i) {
    const auto se = norm(out_reference[i] - out_algorithm[i]);
    mse += se;
  }

  if constexpr (config::print_results) {
    for (int i = 0; i < data.size() * 16; ++i) {
      std::cout << " Ref: " << out_reference[i].real() << " "
                << out_reference[i].imag()
                << " Ten: " << out_algorithm[i].real() << " "
                << out_algorithm[i].imag() << std::endl;
    }
  }

  mse /= data.size();

  std::cout << "Transfers Mode, Time Tensor, Time cuFFTDx, MSE" << std::endl;
  std::cout << "Included," << alg_run_transfers << "," << ref_run_transfers
            << "," << mse << std::endl;
  std::cout << "Excluded," << alg_run_no_transfers << ","
            << ref_run_no_transfers << ","
            << "N/A" << std::endl;
}
