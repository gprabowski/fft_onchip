#include "hip/hip_runtime.h"
#include <algorithm.cuh>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include <hip/hip_fp16.h>

#include <config.hpp>

#include <cute/tensor.hpp>

namespace fft {

__device__ static constexpr float PI = 3.14159265359;

#define gpuErrchk(ans)                                                         \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
            line);
    if (abort)
      exit(code);
  }
}

inline __device__ __half2 hcmul(const __half2 &a, const __half2 &b) {
  return {a.x * b.x - a.y * b.y, a.x * b.y + a.y * b.x};
}

__device__ __host__ constexpr static inline int ilog2(unsigned int n) {
  return 31 - __builtin_clz(n);
}

inline __device__ __half2 exp_alpha(__half alpha) {
  __half2 ret;
  ret.x = hcos(alpha);
  ret.x = hsin(alpha);
  return ret;
}

inline __device__ __half2 pow_theta(int p, int q) {
  p = p % q;
  const auto ang = (__half(-2.f) * __half(PI) * __half(p)) / __half(q);
  return {hcos(ang), hsin(ang)};
}

__device__ int reverseDigits(int number, int base) {
  int reversedNumber = 0;

  while (number > 0) {
    reversedNumber = reversedNumber * base + number % base;
    number /= base;
  }

  return reversedNumber;
}

__global__ void onchip_reference(__half2 *data) {
  extern __shared__ __half2 shared_data[];
  __half2 *shared_F = shared_data + N;

  const auto tid = threadIdx.x;

  // 1. copy data
  for (int id = tid; id < N; id += blockDim.x) {
    shared_data[id] = data[id];
    const auto column = id / 32;
    const auto row = id % 32;
    shared_F[column * row] = pow_theta(row * column, N);
  }
  __syncthreads();

  // perform FFT
  const auto warpIdx = tid / 32;
  const auto laneIdx = tid % 32;
  const auto k = laneIdx;
  // perform two radix-32 iterations
  __half2 local = shared_data[warpIdx + k * 32];
  __half2 result = {__half(0), __half(0)};
  __syncwarp();
// perform warp local butterfly
#define FULL_MASK 0xffffffff
  for (int m = 0; m < 32; ++m) {
    result += hcmul(pow_theta(m * k, 32), __shfl_sync(FULL_MASK, local, m));
  }
  shared_data[warpIdx + k * 32] = result;
  __syncthreads();

  local = shared_data[warpIdx * 32 + k];
  result = __half2{__half(0), __half(0)};
  __syncwarp();
  for (int m = 0; m < 32; ++m) {
    result += hcmul(pow_theta(m * reverseDigits(warpIdx * 32 + k, 32), N),
                    __shfl_sync(FULL_MASK, local, m));
  }

  data[reverseDigits(warpIdx * 32 + k, 32)] = result;
}

size_t run_algorithm(const std::vector<__half2> &data,
                     std::vector<__half2> &out) {
  thrust::host_vector<__half2> h_data;
  thrust::device_vector<__half2> d_data;

  for (int i = 0; i < data.size(); ++i) {
    h_data.push_back({data[i].x, data[i].y});
  }

  d_data = h_data;

  auto t1 = std::chrono::high_resolution_clock::now();
  onchip_reference<<<1, 32, N * 2 * sizeof(__half2)>>>(
      thrust::raw_pointer_cast(d_data.data()));
  hipDeviceSynchronize();
  h_data = d_data;
  auto t2 = std::chrono::high_resolution_clock::now();

  const auto res_ms =
      std::chrono::duration_cast<std::chrono::microseconds>(t2 - t1);

  for (int i = 0; i < data.size(); ++i) {
    out[i] = __half2{h_data[i].x, h_data[i].y};
  }

  return res_ms.count();
}
} // namespace fft
